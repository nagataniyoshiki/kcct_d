#include "hip/hip_runtime.h"
/* DATA_SIZE �̕��������_�̐ω��Z�� CPU ����� GPU �ł����Ȃ� - 2 */
/*  - rev.201905 by Yoshiki NAGATANI */

/*  - CPU ���� OpenMP �ɂ��v�Z�Ƃ���r����
		Visual Studio �ł� [�v���W�F�N�g]-[xx�̃v���p�e�B]-
		[CUDA C/C++]-[Host]-[Additional Compiler Options]
		�Ɂu -Xcompiler "/openmp" �v��ǉ��B
/*  - ���Ԃ��v������i�������b�P�ʁj */

#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <time.h>

/* DATA_SIZE = BLOCK_SIZE * GRID_SIZE �Ŋ���؂�邱��(�v���O�������ł̓m�[�`�F�b�N) */
#define DATA_SIZE 1048576
#define BLOCK_SIZE 256
#define GRID_SIZE (DATA_SIZE/BLOCK_SIZE)

/* ���x��r�̂��ߓ����v�Z�� REPEAT ��J��Ԃ� */
#define REPEAT 10000

/*-----------------------------------------------------------*/
/* CPU���Őω��Z R=A*B �������Ȃ��֐�(�P��R�A) */
void MultiplyOnCPU_Single(float* h_data_A, float* h_data_B, float* h_data_R) {
	long i;

	/* CPU �ł̓f�[�^�̐����� for �����܂킷 */
	for (i = 0; i < DATA_SIZE; i++) {
		h_data_R[i] = h_data_A[i] * h_data_B[i];
	}
}

/*-----------------------------------------------------------*/
/* CPU���Őω��Z R=A*B �������Ȃ��֐�(OpenMP) */
void MultiplyOnCPU_OpenMP(float* h_data_A, float* h_data_B, float* h_data_R) {
	long i;

	/* CPU �ł̓f�[�^�̐����� for �����܂킷 */
	#pragma omp parallel for
	for (i = 0; i < DATA_SIZE; i++) {
		h_data_R[i] = h_data_A[i] * h_data_B[i];
	}
}

/*-----------------------------------------------------------*/
/* GPU���Őω��Z R=A*B �������Ȃ��֐� */
__global__ void MultiplyOnGPU(float* d_data_A, float* d_data_B, float* d_data_R) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	/* GPU �ł� for ���ł͂Ȃ��C�����̒S���̃f�[�^(id)�����v�Z����� OK */
	d_data_R[id] = d_data_A[id] * d_data_B[id];
}

/*-----------------------------------------------------------*/
int main(void) {

	int i;
	time_t time_start_cpu_single, time_end_cpu_single;
	time_t time_start_cpu_openmp, time_end_cpu_openmp;
	time_t time_start_gpu, time_end_gpu;

	printf("DATA_SIZE(%d) = BLOCK_SIZE(%d) x GRID_SIZE(%d).\n", DATA_SIZE, BLOCK_SIZE, GRID_SIZE);

	float* h_data_A;   /* Host(CPU)�������� */
	float* h_data_B;   /* Host(CPU)�������� */
	float* h_data_R;   /* Host(CPU)�������� */
	float* h_data_R_fromGPU;   /* Host(CPU)���������i���ʂ̃`�F�b�N��p�j */

	float* d_data_A;   /* Devive(GPU)�������� */
	float* d_data_B;   /* Devive(GPU)�������� */
	float* d_data_R;   /* Devive(GPU)�������� */

	/* �z�X�g(CPU)���������̈�̊m�ہi�ǐ��d���̂��߃G���[�`�F�b�N�����Ȃ̂Œ��Ӂj */
	h_data_A = (float*)malloc(DATA_SIZE * sizeof(float));
	h_data_B = (float*)malloc(DATA_SIZE * sizeof(float));
	h_data_R = (float*)malloc(DATA_SIZE * sizeof(float));
	h_data_R_fromGPU = (float*)malloc(DATA_SIZE * sizeof(float));

	/* �f�o�C�X(GPU)���������̈�̊m�ہi�ǐ��d���̂��߃G���[�`�F�b�N�����Ȃ̂Œ��Ӂj */
	hipMalloc((void**)& d_data_A, DATA_SIZE * sizeof(float));
	hipMalloc((void**)& d_data_B, DATA_SIZE * sizeof(float));
	hipMalloc((void**)& d_data_R, DATA_SIZE * sizeof(float));

	/* �f�[�^����(���̗�ł�CPU���Ő������Ă���) */
	for (i = 0; i < DATA_SIZE; i++) {
		h_data_A[i] = (double)(rand()) / 32768.0;
		h_data_B[i] = (double)(rand()) / 32768.0;
		h_data_R[i] = 0.0;
	}

	/* �f�o�C�X�Ƀ��������e���R�s�[(CPU��GPU) */
	hipMemcpy(d_data_A, h_data_A, DATA_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_data_B, h_data_B, DATA_SIZE * sizeof(float), hipMemcpyHostToDevice);

	/* �z�X�g(Single CPU)�Őω��Z�����s�i���x�v���̂��� REPEAT ��J��Ԃ��j */
	printf("Start calculation on Single CPU for %d times...", REPEAT);
	time_start_cpu_single = time(NULL);
	for (i = 0; i < REPEAT; i++) {
		MultiplyOnCPU_Single(h_data_A, h_data_B, h_data_R);
	}
	time_end_cpu_single = time(NULL);
	printf("done!! (Time: %d s)\n", time_end_cpu_single - time_start_cpu_single);

	/* �z�X�g(CPU with OpenMP)�Őω��Z�����s�i���x�v���̂��� REPEAT ��J��Ԃ��j */
	printf("Start calculation on CPU with OpenMP for %d times...", REPEAT);
	time_start_cpu_openmp = time(NULL);
	for (i = 0; i < REPEAT; i++) {
		MultiplyOnCPU_OpenMP(h_data_A, h_data_B, h_data_R);
	}
	time_end_cpu_openmp = time(NULL);
	printf("done!! (Time: %d s)\n", time_end_cpu_openmp - time_start_cpu_openmp);

	/* �f�o�C�X(GPU)�Őω��Z�����s�i���x�v���̂��� REPEAT ��J��Ԃ��j */
	printf("Start calculation on GPU for %d times...", REPEAT);
	time_start_gpu = time(NULL);
	for (i = 0; i < REPEAT; i++) {
		MultiplyOnGPU << <GRID_SIZE, BLOCK_SIZE >> > (d_data_A, d_data_B, d_data_R);
	}
	time_end_gpu = time(NULL);
	printf("done!! (Time: %d s)\n", time_end_gpu - time_start_gpu);

	/* �f�o�C�X���烁�������e���R�s�[(CPU��GPU) */
	hipMemcpy(h_data_R_fromGPU, d_data_R, DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	/* ���ʂ̔�r(CPU���)(��ʕ\���̓s����C�ŏ��ƍŌ�̃f�[�^�����\��) */
	printf("Comparison of the Results:\n");
	printf(" %8d: CPU:%f vs GPU:%f\n", 0, h_data_R[0], h_data_R_fromGPU[0]);
	printf(" %8d: CPU:%f vs GPU:%f\n", DATA_SIZE - 1, h_data_R[DATA_SIZE - 1], h_data_R_fromGPU[DATA_SIZE - 1]);

	hipDeviceReset();
	return 0;
}
