/* CPU��GPU�ւ̃f�[�^�R�s�[ �� CPU��GPU�ւ̃f�[�^�R�s�[ */
/*  - rev.201905 by Yoshiki NAGATANI */


#include <hip/hip_runtime.h>
#include <stdio.h>

/* DATA_SIZE = BLOCK_SIZE * GRID_SIZE �Ŋ���؂�邱��(�v���O�������ł̓m�[�`�F�b�N) */
#define DATA_SIZE 8
#define BLOCK_SIZE 4
#define GRID_SIZE (DATA_SIZE/BLOCK_SIZE)

/*-----------------------------------------------------------*/
/* GPU���Ńf�[�^���e��2�{���ĕ\������֐� */
__global__ void DoubleOnGPU(float* d_data, float* d_data2) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	/* GPU �ł� for ���ł͂Ȃ��C�����̒S���̃f�[�^(id)�����v�Z����� OK */
	d_data2[id] = d_data[id] * 2.0;
	printf("My target is d_data[%d] : %f * 2.0 = %f.\n", id, d_data[id], d_data2[id]);
}

/*-----------------------------------------------------------*/
int main(void) {

	float* h_data;    /* Host(CPU)�������� */
	float* h_data2;   /* Host(CPU)�������� */

	float* d_data;    /* Device(GPU)�������� */
	float* d_data2;   /* Device(GPU)�������� */

	/* �z�X�g(CPU)���������̈�̊m�ہi�ǐ��d���̂��߃G���[�`�F�b�N�����Ȃ̂Œ��Ӂj */
	h_data = (float*)malloc(DATA_SIZE * sizeof(float));
	h_data2 = (float*)malloc(DATA_SIZE * sizeof(float));

	/* �f�o�C�X(GPU)���������̈�̊m�ہi�ǐ��d���̂��߃G���[�`�F�b�N�����Ȃ̂Œ��Ӂj */
	hipMalloc((void**)& d_data, DATA_SIZE * sizeof(float));
	hipMalloc((void**)& d_data2, DATA_SIZE * sizeof(float));

	/* �����l�̑��(CPU���Ő���) */
	printf("Data before processing: ");
	for (int i = 0; i < DATA_SIZE; i++) {
		h_data[i] = (float)(i) * 10.0;
		printf("%f, ", h_data[i]);
	}
	printf("\n");

	/* �f�o�C�X�Ƀ��������e���R�s�[(CPU��GPU) */
	hipMemcpy(d_data, h_data, DATA_SIZE * sizeof(float), hipMemcpyHostToDevice);

	/* �f�o�C�X(GPU)��2�{���������s */
	DoubleOnGPU <<<GRID_SIZE, BLOCK_SIZE>>> (d_data, d_data2);

	/* �f�o�C�X���烁�������e���R�s�[(CPU��GPU) */
	hipMemcpy(h_data2, d_data2, DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipDeviceReset();

	/* ���ʂ̕\��(CPU����) */
	printf("Data after processing: ");
	for (int i = 0; i < DATA_SIZE; i++) {
		printf("%f, ", h_data2[i]);
	}

	return 0;
}
